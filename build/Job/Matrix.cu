#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

// 30 * 30的矩阵

// 随机初始化数组
void initialInt(float *ip, float size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xff) / 66.6;
	}
}
// 打印数组
void printMatrix(float *A, float *B, float *C, const int nx, const int ny)
{
	float *ia = A, *ib = B, *ic = C;
	printf("\nMatrix:(%d, %d)\n", nx, ny);
	for (int iy = 0; iy < ny; iy++)
	{
		for (int ix = 0; ix < nx; ix++)
		{
			printf("%f + %f = %f     ", ia[ix], ib[ix], ic[ix]);
		}
		ia += nx;
		ib += nx;
		ic += nx;
		printf("\n");
	}
	printf("\n");
}
// 验证结果
void printResult(float *C, float *CC, const int nx, const int ny)
{
	float *ic = C, *icc = CC;
	for (int iy = 0; iy < ny; iy++)
	{
		for (int ix = 0; ix < nx; ix++)
		{
			printf("%f     ", ic[ix]-icc[ix]);
		}
		ic += nx;
		icc += nx;
		printf("\n");
	}
	printf("\n");
}

// GPU：计算C=A+B
__global__ void sumMatrixOnDevice(float *MatA, float *MatB, float *MatC, const int nx, const int ny)
{
	int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int iy = threadIdx.y + blockDim.y*blockIdx.y;
	unsigned int idx = iy * nx + ix;
	//unsigned int t_n = gridDim.x*blockDim.x + gridDim.y*blockDim.y;
	if (ix < nx && iy < ny)
	{
		MatC[idx] = MatA[idx] + MatB[idx];
	}
}

// GPU：计算C=A*B
__global__ void MatMul(float *M,float *N,float *P,int width)
{
	int x = threadIdx.x;
	int y = threadIdx.y;

	float Pervalue = 0;

	float elem1 = 0.0,elem2 = 0.0,value = 0.0;
	for(int i = 0;i < width;i++)
	{
		elem1 = M[y * width + i];//取M矩阵的一行
		elem2 = N[i * width + x];//取N矩阵的一列

		value += elem1 * elem2;//求和
	}

	P[y * width + x] = value;
}


int main(int argc, char **argv)
{
	//printf("%s Starting...\n", argv[10]);

	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("Using Device  %d: %s\n\n", dev, deviceProp.name);

	// set matrix dimension
	int nx = 30;
	int ny = 30;
	int nxy = nx * ny;
	int nBytes = nxy * sizeof(float);

	// malloc host dimension
	float *h_A, *h_B, *h_C, *h_CC;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	h_C = (float *)malloc(nBytes);
	h_CC = (float *)malloc(nBytes);

	// initialize host matrix with integer
	initialInt(h_A, nxy);
	initialInt(h_B, nxy);

	// 开始计时
	clock_t cpuStart = clock();

	sumMatrixOnHost(h_A, h_B, h_C, nx, ny);

	// 结束计时
	clock_t cpuEnd = clock();
	float cpuTime = (float)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;
	printf("cpu time:%f\n", cpuTime);

	// mallox device memory
	float *d_MatA, *d_MatB, *d_MatC;
	hipMalloc((void **)&d_MatA, nBytes);
	hipMalloc((void **)&d_MatB, nBytes);
	hipMalloc((void **)&d_MatC, nBytes);

	// 开始计时
// 	clock_t gpuStart = clock();

	// transfer data from host to device
	hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

	//set up execution configuration
	int dimx = 32;
	int dimy = 32;
	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);


	// 矩阵加法
	sumMatrixOnDevice << <grid, block >> > (d_MatA, d_MatB, d_MatC, nx, ny);
	hipDeviceSynchronize();
	// transfer data from device to host
	hipMemcpy(h_CC, d_MatC, nBytes, hipMemcpyDeviceToHost);
	printResult(h_C, h_CC, nx, ny);

	// 矩阵乘法
	MatMul<<<1,blockSize>>>(d_MatA,d_MatB,d_MatC,nx);//调用核函数
	hipDeviceSynchronize();
	hipMemcpy(h_CC,d_MatC,nBytes,hipMemcpyDeviceToHost);
    printf("c0 = %d \n",h_CC[0][0]);


	// 结束计时
    // 	clock_t gpuEnd = clock();
    // 	float gpuTime = (float)(gpuEnd - gpuStart) / CLOCKS_PER_SEC;
    // 	printf("gpu time:%f\n", gpuTime);

	// free host and device memory
	hipFree(d_MatA);
	hipFree(d_MatB);
	hipFree(d_MatC);
	free(h_A);
	free(h_B);
	free(h_C);

	// reset device
	hipDeviceReset();

	return 0;
}