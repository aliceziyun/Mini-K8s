#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define THREAD_NUM 256

#define MATRIX_SIZE 10

const int blocks_num = (MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

// CUDA 初始化
bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0)
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for (i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
            if (prop.major >= 1)
            {
                break;
            }
        }
    }

    if (i == count)
    {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);
    return true;
}

void generateMatrix(int *a, int size)
{
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            a[i * size + j] = rand() % 256;
        }
    }
}

void printMatrix(int *a, int size)
{
    //print
    // puts("===========Print a Matrix===========");
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%d ",a[i * size + j]);
        }
        puts("");
    }
    puts("");
}

__global__ static void addMatrixCUDA(const int *a, const int *b, int *c, int size)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int idx = bid * THREAD_NUM + tid;

    if (idx < size)
    {
        int max = size * size;
        for (int i = idx; i < max; i += size) {
            c[i] = a[i] + b[i];
        }
    }
}

__global__ static void multiMatrixCUDA(const int *a, const int *b, int *c, int size)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int idx = bid * THREAD_NUM + tid;
    const int row = idx / size;
    const int column = idx % size;

    if (row < size && column < size)
    {
        int t = 0;

        for (int i = 0; i < size; i++)
        {
            t += a[row * size + i] * b[i * size + column];
        }
        c[row * size + column] = t;
    }
}

int main()
{
    if (!InitCUDA())
        return 0;

    srand(0);

    int *a, *b, *c, *d;
    a = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    b = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    c = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    d = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);

    generateMatrix(a, MATRIX_SIZE);
    generateMatrix(b, MATRIX_SIZE);

    //print a, b
    puts("[Matrix a]:");
    printMatrix(a, MATRIX_SIZE);
    puts("[Matrix b]:");
    printMatrix(b, MATRIX_SIZE);

    int *cuda_a, *cuda_b, *cuda_c, *cuda_d;

    hipMalloc((void **)&cuda_a, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    hipMalloc((void **)&cuda_b, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    hipMalloc((void **)&cuda_c, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    hipMalloc((void **)&cuda_d, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);

    hipMemcpy(cuda_a, a, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);

    //加法
    addMatrixCUDA <<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c, MATRIX_SIZE);
    hipMemcpy(c, cuda_c, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost);
    //print c
    puts("[Matrix c]:");
    printMatrix(c, MATRIX_SIZE);

    //乘法
    multiMatrixCUDA <<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_d, MATRIX_SIZE);
    hipMemcpy(d, cuda_d, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost);
    //print d
    puts("[Matrix d]:");
    printMatrix(d, MATRIX_SIZE);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
    hipFree(cuda_d);

    return 0;
}